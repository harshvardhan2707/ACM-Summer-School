
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <unistd.h>

#define N 11

__global__ void MatAdd(int A[][N], int B[][N], int C[][N]){
           int i = threadIdx.x;
           int j = threadIdx.y;

           C[i][j] = A[i][j]+B[i][j];
       }


int main(){

    int A[N][N];
    int B[N][N];
    int C[N][N];

    for(int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
                A[i][j] = 1;
                B[i][j] = 1;
                C[i][j] = 99;
        }
    }

    int (*pA)[N], (*pB)[N], (*pC)[N];

    hipMalloc((void**)&pA, (N*N)*sizeof(int));
    hipMalloc((void**)&pB, (N*N)*sizeof(int));
    hipMalloc((void**)&pC, (N*N)*sizeof(int));

    hipMemcpy(pA, A, (N*N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pB, B, (N*N)*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pC, C, (N*N)*sizeof(int), hipMemcpyHostToDevice);

    struct timeval tv1,tv2;
    struct timezone tz;
    double elapsed;

    int numBlocks = 1;
    dim3 threadsPerBlock(N,N);

    gettimeofday(&tv1,&tz);

    MatAdd<<<numBlocks,threadsPerBlock>>>(pA,pB,pC);

    gettimeofday(&tv2,&tz);
    elapsed = (double) (tv2.tv_sec - tv1.tv_sec) + (double) (tv2.tv_usec - tv1.tv_usec)*1.e-6;
    printf("Elapsed time=%4.2lf seconds.\n",elapsed);

    hipMemcpy(C, pC, (N*N)*sizeof(int), hipMemcpyDeviceToHost);

    int i, j; printf("C = \n");
    for(i=0;i<N;i++){
        for(j=0;j<N;j++){
            printf("%d ", C[i][j]);
        }
        printf("\n");
    }

    hipFree(pA);
    hipFree(pB);
    hipFree(pC);

    printf("\n");

    return 0;
}
                                           

